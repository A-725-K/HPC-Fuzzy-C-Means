#include "hip/hip_runtime.h"
#include "../cuda/fuzzy_functions.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include ""
#include <chrono>


#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


__host__ __device__ double distance(point a, point b) {
    double sum = 0;
    for (int k = 0; k < DIM; k++) {
        sum += pow(a.dims[k] - b.dims[k], 2);
    }

    return sqrt(sum);
}

int bestClusterIndex(double *v) {
    int idxMax = 0;
    for (int i=1; i<N_CL; i++)
        if (v[i] > v[idxMax])
            idxMax = i;
    return idxMax;
}

double fuzzy(point x, point *clusters_centers, int clusterIdx) {
    double sum = 0.0;
    for (int i=0; i<N_CL; i++) {
        double frac = distance(x, clusters_centers[clusterIdx]) / distance(x, clusters_centers[i]);
        sum += pow(frac, 2/(M-1));
    }
    return 1/sum;
}

void initializeClustersCenters(point *cc, point *X) {
    for (int i=0; i<N_CL; i++) {
        int r = (rand() % N_POINTS) + i*N_POINTS;
        for(int k=0; k<DIM; k++)
            cc[i].dims[k] = X[r].dims[k];
    }
}

void adjustClustersCenters(point* cc, double* mv, point* X) {
    for (int k = 0; k < N_CL; k++) {
        double sum_dims[DIM];
        for (int i = 0; i < DIM; i++)
            sum_dims[i] = 0.0;

        double sum_den = 0.0;
        for (int i = 0; i < SIZE; i++) {
            double u_ij_m = pow(mv[i * N_CL + k], M);
            for (int c = 0; c < DIM; c++)
                sum_dims[c] += u_ij_m * X[i].dims[c];
            sum_den += u_ij_m;
        }

        for (int v = 0; v < DIM; v++)
            cc[k].dims[v] = sum_dims[v] / sum_den;
    }
}

void initializeMembershipVecs(double *mv, point *cc, point *X) {
    for (int i=0; i<SIZE; i++)
        for (int j=0; j<N_CL; j++)
            mv[i*N_CL + j] = distance(X[i], cc[j]);
}

__global__ void objectiveFunction(double *mv, point *X, point *cc, double *J) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < SIZE && col < N_CL)
        *J += pow(mv[row*N_CL + col], M) * pow(distance(X[row], cc[col]), 2);
}

void fuzzyCMeans(point *X, int *Y) {
    double *membership_vecs = new double[SIZE*N_CL];
    point* clusters_centers = new point[N_CL];

    int n;
    double J, old_J = 0.0;

    initializeClustersCenters(clusters_centers, X);
    initializeMembershipVecs(membership_vecs, clusters_centers, X);

    const int THREADS = 32;
    const dim3 threads(THREADS, THREADS);
    const int BLOCKS = ((SIZE + threads.x - 1) / threads.x);
    const dim3 blocks(BLOCKS, BLOCKS);

    point *GPU_X, *GPU_cluster_centers;
    double* GPU_membership_vecs, *GPU_oldJ, *GPU_J;

    size_t dimX = SIZE * sizeof(point);
    size_t dimCC = N_CL * sizeof(point);
    size_t dimMV = SIZE * N_CL * sizeof(double);
    size_t dimD = sizeof(double);

    hipMalloc(&GPU_X, dimX);
    hipMalloc(&GPU_cluster_centers, dimCC);
    hipMalloc(&GPU_membership_vecs, dimMV);
    hipMalloc(&GPU_J, dimD);
    hipMalloc(&GPU_oldJ, dimD);

    hipMemcpy(GPU_X, X, dimX, hipMemcpyHostToDevice);
    hipMemcpy(GPU_cluster_centers, clusters_centers, dimCC, hipMemcpyHostToDevice);
    hipMemcpy(GPU_membership_vecs, membership_vecs, dimMV, hipMemcpyHostToDevice);
    hipMemcpy(GPU_J, &J, dimD, hipMemcpyHostToDevice);
    hipMemcpy(GPU_oldJ, &old_J, dimD, hipMemcpyHostToDevice);

    objectiveFunction<<< blocks, threads >>>(GPU_membership_vecs, GPU_X, GPU_cluster_centers, GPU_oldJ);

    hipDeviceSynchronize();
    hipMemcpy(&old_J, GPU_oldJ, dimD, hipMemcpyDeviceToHost);

    for (n=0; n<N_ITER; n++) {
        adjustClustersCenters(clusters_centers, membership_vecs, X);

        for (int i=0; i<SIZE; i++)
            for (int j=0; j<N_CL; j++)
                membership_vecs[i*N_CL + j] = fuzzy(X[i], clusters_centers, j);
    
        hipMemcpy(GPU_cluster_centers, clusters_centers, dimCC, hipMemcpyHostToDevice);
        hipMemcpy(GPU_membership_vecs, membership_vecs, dimMV, hipMemcpyHostToDevice);

        objectiveFunction<<< blocks, threads >>>(GPU_membership_vecs, GPU_X, GPU_cluster_centers, GPU_J);
        
        hipDeviceSynchronize();
        hipMemcpy(&J, GPU_J, dimD, hipMemcpyDeviceToHost);

        if (fabs(old_J - J) < EPS)
            break;
        old_J = J;
    }

    for (int i=0; i<SIZE; i++)
        Y[i] = bestClusterIndex(membership_vecs + i*N_CL);
    
    std::cout << "n:" << n << std::endl;

    hipFree(GPU_X);
    hipFree(GPU_cluster_centers);
    hipFree(GPU_membership_vecs);
    hipFree(GPU_J);
    hipFree(GPU_oldJ);

    delete[] membership_vecs;
    delete[] clusters_centers;
}

